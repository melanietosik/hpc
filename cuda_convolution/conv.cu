#include "hip/hip_runtime.h"
#include <hipDNN.h>
#include <iostream>
#include <stdexcept>
#include <stdio.h>

#define H 4096
#define W 4096
#define C 3
#define FH 3
#define FW 3
#define K 10
#define P 1   // Padding
#define N 16  // Block size

#define I_size (C * W * H)
#define F_size (K * C * FH * FW)
#define O_size (K * W * H)

#define idx3(c,x,y) ((c)*(H*W) + (x)*W + (y))
#define idx4(k,c,i,j) (k*(C*FH*FW) + c*(FH*FW) + (i)*FW + (j))  // Brackets are life savers

#define CHECK(expression)                                  \
{                                                          \
  hipdnnStatus_t status = (expression);                     \
  if (status != HIPDNN_STATUS_SUCCESS) {                    \
    std::cerr << "Error on line " << __LINE__ << ": "      \
              << hipdnnGetErrorString(status) << std::endl; \
    std::exit(EXIT_FAILURE);                               \
  }                                                        \
}

/*
  Usage (on HPC):

    module purge
    module load cuda/9.0.176
    module load cudnn/9.0v7.0.5
    make && ./conv
*/

__global__ void convolutionKernel(double* I, double* F, double* O) {

  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z;

  // Skip computation if thread is outside boundaries
  if (k >= K || col >= W || row >= H) return;
    
  int x = col - P; // x and y correspond to I0 dimensions
  int y = row - P;

  double pixel = 0.0;

  for (int c=0; c<C; c++) {
    for (int j=0; j<FH; j++) {
      for (int i=0; i<FW; i++) {

        int cur_row = x + i;
        int cur_col = y + j;

        // Implicit handling of I0 padding
        double I0_val;
        if (cur_row == -1 || cur_row == W || cur_col == -1 || cur_col == H)
          I0_val = 0;
        else
          I0_val = I[idx3(c, cur_row, cur_col)];  // Index lookup

        // Compute and update output value
        pixel += (F[idx4(k, c, FW - 1 - i, FH - 1 - j)] * I0_val);
      }
    }
  }
  // Add computed pixel value to output
  O[idx3(k,col,row)] = pixel;
}

double* init_I(double* I) {
  /*
    Initialize input tensor I
    :: I[c, x, y] = c * (x + y)
  */
  int c, x, y;
  for (c=0; c<C; c++) {
    for (x=0; x<H; x++) {
      for (y=0; y<W; y++) {
        I[idx3(c, x, y)] = c * (x + y); 
  }}}
  return I;
}

double* init_F(double* F) {
  /*
    Initialize filter tensor I
    :: F[k, c, i, j] = (c + k) * (i + j);
  */
  int k, c, i, j;
  for (k=0; k<K; k++) {
    for (c=0; c<C; c++) {
      for (i=0; i<FW; i++) {
        for (j=0; j<FH; j++) {
          F[idx4(k,c,i,j)] = (c + k) * (i + j);
  }}}}
  return F;
}

void cuDNNConvolution(double* d_I, double* d_F, double* d_O) {
  /*
    Convolution in cuDNN
    cf. http://www.goldsborough.me/cuda/ml/cudnn/c++/
        2017/10/01/14-37-23-convolutions_with_cudnn/
  */
  hipdnnHandle_t cudnn;
  CHECK(hipdnnCreate(&cudnn));

  // I
  hipdnnTensorDescriptor_t I_descriptor;
  CHECK(hipdnnCreateTensorDescriptor(&I_descriptor));
  CHECK(hipdnnSetTensor4dDescriptor(I_descriptor,
    HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, C, H, W));

  // F
  hipdnnFilterDescriptor_t F_descriptor;
  CHECK(hipdnnCreateFilterDescriptor(&F_descriptor));
  CHECK(hipdnnSetFilter4dDescriptor(F_descriptor,
    HIPDNN_DATA_DOUBLE, HIPDNN_TENSOR_NCHW, K, C, FH, FW));

  // O
  hipdnnTensorDescriptor_t O_descriptor;
  CHECK(hipdnnCreateTensorDescriptor(&O_descriptor));
  CHECK(hipdnnSetTensor4dDescriptor(O_descriptor,
    HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, K, H, W));

  // C
  hipdnnConvolutionDescriptor_t CONV_descriptor;
  CHECK(hipdnnCreateConvolutionDescriptor(&CONV_descriptor));
  CHECK(hipdnnSetConvolution2dDescriptor(CONV_descriptor,
    P,  // Padding height
    P,  // Padding width
    1,  // Vertical stride
    1,  // Horizontal stride
    1,  // Dilation height
    1,  // Dilation width
    // HIPDNN_CROSS_CORRELATION,
    HIPDNN_CONVOLUTION,
    HIPDNN_DATA_DOUBLE));

  // Convolution algorithm
  hipdnnConvolutionFwdAlgo_t CONV_algorithm;
  CHECK(hipdnnGetConvolutionForwardAlgorithm(cudnn,
    I_descriptor, F_descriptor, CONV_descriptor, O_descriptor,
    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
    0, &CONV_algorithm));

  // Workspace specification
  size_t workspace_bytes = 0;
  CHECK(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
    I_descriptor, F_descriptor, CONV_descriptor, O_descriptor,
    CONV_algorithm, &workspace_bytes));

  void* d_w;
  hipMalloc(&d_w, sizeof(double)*workspace_bytes);

  double alpha = 1.;
  double beta = 0.;  // Not ResNet

  // Compute convolution
  CHECK(hipdnnConvolutionForward(cudnn,
    &alpha,
    I_descriptor, d_I,
    F_descriptor, d_F,
    CONV_descriptor,
    CONV_algorithm,
    d_w,
    workspace_bytes,
    &beta,
    O_descriptor, d_O));

  hipFree(d_w);
  CHECK(hipdnnDestroyTensorDescriptor(I_descriptor));
  CHECK(hipdnnDestroyTensorDescriptor(O_descriptor));
  CHECK(hipdnnDestroyFilterDescriptor(F_descriptor));
  CHECK(hipdnnDestroyConvolutionDescriptor(CONV_descriptor));
  CHECK(hipdnnDestroy(cudnn));
}

int main(void) {
  /*
    Main function
  */
  double* I = (double*) malloc(I_size * sizeof(double));
  double* F = (double*) malloc(F_size * sizeof(double));
  double* O = (double*) malloc(O_size * sizeof(double));
  init_I(I);
  init_F(F);
  double* d_I;  // Device I
  double* d_F;  // Device F
  double* d_O;  // Device O

  hipError_t malloc_I = hipMalloc(&d_I, I_size * sizeof(double)); 
  hipError_t malloc_F = hipMalloc(&d_F, F_size * sizeof(double));
  hipError_t malloc_O = hipMalloc(&d_O, O_size * sizeof(double));
  if (malloc_I != hipSuccess) throw std::runtime_error("Failed to allocate memory: I");
  if (malloc_F != hipSuccess) throw std::runtime_error("Failed to allocate memory: F");
  if (malloc_O != hipSuccess) throw std::runtime_error("Failed to allocate memory: O");

  hipError_t cp_I = hipMemcpy(d_I, I, I_size * sizeof(double), hipMemcpyHostToDevice);
  hipError_t cp_F = hipMemcpy(d_F, F, F_size * sizeof(double), hipMemcpyHostToDevice);
  hipError_t cp_O = hipMemcpy(d_O, O, O_size * sizeof(double), hipMemcpyHostToDevice);
  if (cp_I != hipSuccess) throw std::runtime_error("Failed to copy to host memory: I");
  if (cp_F != hipSuccess) throw std::runtime_error("Failed to copy to host memory: F");
  if (cp_O != hipSuccess) throw std::runtime_error("Failed to copy to host memory: O");

  /* Accounting
     cf. https://devblogs.nvidia.com/how-implement-performance-metrics-cuda-cc/
  */
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  double checksum;
  float milliseconds;

  /* CUDA simple convolution kernel */
  dim3 threadsPerBlock(N, N);
  dim3 numBlocks(W/threadsPerBlock.x, H/threadsPerBlock.y, K);

  hipEventRecord(start);  // Start time
  convolutionKernel<<<numBlocks, threadsPerBlock>>>(d_I, d_F, d_O);
  hipEventRecord(stop);  // End time

  hipMemcpy(O, d_O, O_size * sizeof(double), hipMemcpyDeviceToHost);

  // Get execution time
  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  // Compute checksum
  checksum = 0;
  for (int i=0; i<O_size; i++) {
    checksum += O[i];
  }
  printf("%.2f,%4.3lf\n", checksum, milliseconds);

  /* cuDNN convolution */
  memset(O, 0, O_size * sizeof(double));  // Zero-out O

  cp_O = hipMemcpy(d_O, O, O_size * sizeof(double), hipMemcpyHostToDevice);
  if (malloc_O != hipSuccess) throw std::runtime_error("Failed to allocate memory: O");
  if (cp_O != hipSuccess) throw std::runtime_error("Failed to copy to host memory: O");

  hipEventRecord(start);  // Start time
  cuDNNConvolution(d_I, d_F, d_O);
  hipEventRecord(stop);  // End time

  hipMemcpy(O, d_O, O_size * sizeof(double), hipMemcpyDeviceToHost);

  // Get execution time
  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  // Compute checksum
  checksum = 0;
  for (int i=0; i<O_size; i++) {
    checksum += O[i];
  }
  printf("%.2f,%4.3lf\n", checksum, milliseconds);

  hipFree(d_I);
  hipFree(d_F);
  hipFree(d_O);
}

/*
228686907676500.00,175.743
228686907676500.00,849.415
*/
